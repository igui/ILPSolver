#include "hip/hip_runtime.h"
/*
* Copyright (c) 2015 Opposite Renderer
* For the full copyright and license information, please view the LICENSE.txt
* file that was distributed with this source code.
*/

#include "config.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <optix_world.h>
#include <thrust/reduce.h>
#include <thrust/pair.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/partition.h>
#include <thrust/scan.h>
#include <thrust/adjacent_difference.h>
#include "renderer/ppm/Photon.h"
#include <cstdio>
#include <cmath>
#include "renderer/ppm/PhotonGrid.h"
#include "renderer/Hitpoint.h"
#include "renderer/PPMOptixRenderer.h"
#include "renderer/PMOptixRenderer.h"
#include "util/sutil.h"
#include "renderer/OptixEntryPoint.h"
#include "renderer/helpers/optix.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/nsight.h"
#include "math/Vector3.h"

// From https://devtalk.nvidia.com/default/topic/458062/atomicadd-float-float-atomicmul-float-float-/
__device__ inline void floatAtomicAdd(float* address, float value)
{
	float old = value;
	float new_old;

	do
	{
		new_old = atomicExch(address, 0.0f);
		new_old += old;
	} while ((old = atomicExch(address, new_old)) != 0.0f);
};

__global__ void sumPhotonsHitCount(Photon* photons, unsigned int numPhotons, unsigned int *hitCount, float *rawRadiance)
{
	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < numPhotons)
	{
		Photon & photon = photons[index];
		if (fmaxf(photon.power) > 0)
		{
			atomicAdd(hitCount + photon.objectId, 1);
			floatAtomicAdd(rawRadiance + photon.objectId, photon.power.x + photon.power.y + photon.power.z);
		}
	}
}

void PMOptixRenderer::countHitCountPerObject()
{
	nvtxRangePushA("countHitCountPerObject");
	int deviceNumber = 0;
	hipSetDevice(m_optixDeviceOrdinal);

	thrust::device_ptr<unsigned int> hitCount = getThrustDevicePtr<unsigned int>(m_hitCountBuffer, deviceNumber);
	thrust::fill(hitCount, hitCount + m_sceneObjects, 0);

	thrust::device_ptr<float> rawRadiance = getThrustDevicePtr<float>(m_rawRadianceBuffer, deviceNumber);
	thrust::fill(rawRadiance, rawRadiance + m_sceneObjects, 0);

	unsigned int numPhotons = getNumPhotons();
	const unsigned int blockSize = 512;
	unsigned int numBlocks = numPhotons / blockSize + (numPhotons%blockSize == 0 ? 0 : 1);

	// Get a device_ptr to our photon list
	thrust::device_ptr<Photon> photons = getThrustDevicePtr<Photon>(m_photons, deviceNumber);
	Photon* photonsPtr = thrust::raw_pointer_cast(&photons[0]);
	unsigned int *hitCountPtr = thrust::raw_pointer_cast(&hitCount[0]);
	float *rawRadiancePtr = thrust::raw_pointer_cast(&rawRadiance[0]);

	sumPhotonsHitCount << <numBlocks, blockSize >> > (photonsPtr, numPhotons, hitCountPtr, rawRadiancePtr);
	hipDeviceSynchronize();

	nvtxRangePop();
}