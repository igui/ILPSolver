#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2014 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_cuda.h>
#include <hip/hip_runtime.h>
#include "config.h"
#include "renderer/Light.h"
#include "renderer/ShadowPRD.h"
#include "renderer/RayType.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/random.h"
#include "renderer/ppm/Photon.h"
#include "renderer/ppm/PhotonPRD.h"
#include "math/Sphere.h"

using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtBuffer<Photon, 1> photons;
rtBuffer<RandomState, 2> randomStates;
rtDeclareVariable(uint, maxPhotonDepositsPerEmitted, , );
rtDeclareVariable(uint, photonLaunchWidth, , );
rtBuffer<Light, 1> lights;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(Sphere, sceneBoundingSphere, , );
rtBuffer<float> powerEmitted;

// From https://devtalk.nvidia.com/default/topic/458062/atomicadd-float-float-atomicmul-float-float-/
__device__ inline void floatAtomicAdd(float* address, float value)
{
	float old = value;
	float new_old;

	do
	{
		new_old = atomicExch(address, 0.0f);
		new_old += old;
	}while ((old = atomicExch(address, new_old))!=0.0f);
};


static __device__ void generatePhotonOriginAndDirection(const Light& light, RandomState& state, const Sphere & boundingSphere, 
    float3& origin, float3& direction, float& photonPowerFactor)
{
    origin = light.position;
    float2 sample1 = getRandomUniformFloat2(&state);

    if(light.lightType == Light::AREA)
    {
        float2 sample2 = getRandomUniformFloat2(&state);
        origin += sample1.x*(optix::float3)light.v1 + sample1.y*(optix::float3)light.v2;
        direction = sampleUnitHemisphere(light.normal, sample2);
    }
    else if(light.lightType == Light::POINT)
    {
        // If the point light is well outside the bounding sphere, we make sure to emit 
        // only at the scene (to emulate a directional light)
        float3 sceneCenterToLight = light.position-boundingSphere.center;
        float lightDistance = length(sceneCenterToLight);
        sceneCenterToLight /= lightDistance;
        bool lightWellOutsideSphere = (lightDistance > 1.5*boundingSphere.radius);
        // If light is far away, send photons at the scene and reduce the power based on the solid angle of the scene bounding sphere
        if(lightWellOutsideSphere)
        {
            float3 pointOnDisc = sampleDisc(sample1, boundingSphere.center, boundingSphere.radius, sceneCenterToLight);
            direction = normalize(pointOnDisc-origin);
            // Solid angle of sample disc calculated with http://planetmath.org/calculatingthesolidangleofdisc
            photonPowerFactor = (1  - lightDistance * rsqrtf(boundingSphere.radius*boundingSphere.radius+lightDistance*lightDistance)) / 2.f;
        }
        else
        {
            direction = sampleUnitSphere(sample1);
        }
    }
    else if(light.lightType == Light::SPOT)
    {
        float3 pointOnDisc = sampleDisc(sample1, origin+light.direction, sinf(light.angle/2), light.direction);
        direction = normalize(pointOnDisc-origin);
    }
	else if(light.lightType == Light::DIRECTIONAL)
	{
		direction = light.direction;

		// Good enough (~0.13)
		// origin = sampleUnitSphere(sample1) * sceneBoundingSphere.radius + sceneBoundingSphere.center;
		// photonPowerFactor = 1;
		
		// Too few photons (~0.00)
		// origin = sampleUnitSphere(sample1) * sceneBoundingSphere.radius * 10 + sceneBoundingSphere.center;
		// photonPowerFactor = 10;

		// Good enough (~0.25)
		// origin = sampleDisc(sample1, (float3)sceneBoundingSphere.center - sceneBoundingSphere.radius * direction, sceneBoundingSphere.radius, direction);
		// photonPowerFactor = 1;

		// Test
		origin = sampleDisc(sample1, (float3)sceneBoundingSphere.center - sceneBoundingSphere.radius * direction, sceneBoundingSphere.radius, direction);
		photonPowerFactor = 1;
	}
}

RT_PROGRAM void generator()
{
    PhotonPRD photonPrd;
    photonPrd.pm_index = (launchIndex.y * photonLaunchWidth + launchIndex.x)*maxPhotonDepositsPerEmitted;
    photonPrd.numStoredPhotons = 0;
    photonPrd.depth = 0;
    photonPrd.weight = 1.0f;
    photonPrd.randomState = randomStates[launchIndex];
	photonPrd.inHole = false;

    int lightIndex = 0;
    if(lights.size() > 1)
    {
        float sample = getRandomUniformFloat(&photonPrd.randomState);
        lightIndex = intmin((int)(sample*lights.size()), lights.size()-1);
    }

    Light light = lights[lightIndex];
    float powerScale = lights.size();

    photonPrd.power = light.power*powerScale;

	floatAtomicAdd(&powerEmitted[0], photonPrd.power.x + photonPrd.power.y + photonPrd.power.y);

    float3 rayOrigin, rayDirection;
   
    float photonPowerFactor = 1.f;
    generatePhotonOriginAndDirection(light, photonPrd.randomState, sceneBoundingSphere, rayOrigin, rayDirection, photonPowerFactor);
    photonPrd.power *= photonPowerFactor;


    Ray photon = Ray(rayOrigin, rayDirection, RayType::PHOTON, 0.0001, RT_DEFAULT_MAX );

#if ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_KD_TREE_CPU || ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_UNIFORM_GRID
    // Clear photons owned by this thread
    for(unsigned int i = 0; i < maxPhotonDepositsPerEmitted; ++i)
    {
        photons[photonPrd.pm_index+i].position = make_float3(0.0f);
        photons[photonPrd.pm_index+i].power = make_float3(0.0f);
    }
#endif

    rtTrace( sceneRootObject, photon, photonPrd );

    randomStates[launchIndex] = photonPrd.randomState;


}

rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
RT_PROGRAM void miss()
{
    OPTIX_DEBUG_PRINT(photonPrd.depth, "Photon missed geometry.\n");
}

//
// Exception handler program
//

rtDeclareVariable(float3, exceptionErrorColor, , );
RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
    printf("Exception Photon: %d!\n", code);
	rtPrintExceptionDetails();
    photonPrd.power = make_float3(0,0,0);
}