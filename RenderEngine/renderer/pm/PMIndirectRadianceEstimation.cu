#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

//#define MAX_DEPTH 20

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "renderer/Light.h"
#include "renderer/ppm/Photon.h"
#include "renderer/RayType.h"
#include "renderer/Hitpoint.h"
#include "renderer/ppm/PhotonGrid.h"
#include "renderer/RadiancePRD.h"

using namespace optix;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );

rtBuffer<Photon, 1> photons;
rtBuffer<Hitpoint, 2> raytracePassOutputBuffer;
rtBuffer<float3, 2> indirectRadianceBuffer;

rtDeclareVariable(float, alpha, , );
rtDeclareVariable(float, emittedPhotonsPerIterationFloat, , );
rtDeclareVariable(float, ppmRadius, ,);
rtDeclareVariable(float, ppmRadiusSquared, ,);
rtDeclareVariable(float, ppmRadiusSquaredNew, ,);

rtDeclareVariable(uint3, photonsGridSize, , );
rtDeclareVariable(float3, photonsWorldOrigo, ,);
rtDeclareVariable(float, photonsGridCellSize, ,);
rtBuffer<uint, 1> hashmapOffsetTable;


__device__ __inline float validPhoton(const Photon & photon, const float distance2, const float radius2, const float3 & hitNormal)
{
    return distance2 <= radius2 && dot(-photon.rayDirection, hitNormal) >= 0; 
}

__device__ __inline float3 photonPower(const Photon & photon, const float distance2, const float radius2)
{
    // Use the gaussian filter from Realistic Image Synthesis Using Photon Mapping, Wann Jensen
    const float alpha = 1.818;
    const float beta = 1.953;
    const float expNegativeBeta = 0.141847;
    float weight = alpha*(1 - (1-exp(-beta*distance2/(2*radius2)))/(1-expNegativeBeta));
    return photon.power*weight;
}

RT_PROGRAM void kernel()
{
    Hitpoint rec = raytracePassOutputBuffer[launchIndex];
    
    float3 indirectAccumulatedPower = make_float3( 0.0f, 0.0f, 0.0f );

    int _dPhotonsVisited = 0;
    int _dCellsVisited = 0;

    if(rec.flags & PRD_HIT_NON_SPECULAR)
    {
        float radius2 = ppmRadiusSquared;
        float radius = ppmRadius;
        
        float invCellSize = 1.f/photonsGridCellSize;
        float3 normalizedPosition = rec.position - photonsWorldOrigo;
        unsigned int x_lo = (unsigned int)max(0, (int)((normalizedPosition.x - radius) * invCellSize));
        unsigned int y_lo = (unsigned int)max(0, (int)((normalizedPosition.y - radius) * invCellSize));
        unsigned int z_lo = (unsigned int)max(0, (int)((normalizedPosition.z - radius) * invCellSize));
     
        unsigned int x_hi = (unsigned int)min(photonsGridSize.x-1, (unsigned int)((normalizedPosition.x + radius) * invCellSize));
        unsigned int y_hi = (unsigned int)min(photonsGridSize.y-1, (unsigned int)((normalizedPosition.y + radius) * invCellSize));
        unsigned int z_hi = (unsigned int)min(photonsGridSize.z-1, (unsigned int)((normalizedPosition.z + radius) * invCellSize));    

        if(x_lo <= x_hi)
        {
            for(unsigned int z = z_lo; z <= z_hi; z++)
            {
                for(unsigned int y = y_lo; y <= y_hi; y++)
                {
                    optix::uint3 cell;
                    cell.x = x_lo;
                    cell.y = y;
                    cell.z = z;
                    unsigned int from = getPhotonGridIndex1D(cell, photonsGridSize);
                    unsigned int to = from + (x_hi-x_lo);

                    unsigned int offset = hashmapOffsetTable[from];
                    unsigned int offsetTo = hashmapOffsetTable[to+1];
                    unsigned int numPhotons = offsetTo-offset;

                    _dCellsVisited++;

                    for(unsigned int i = offset; i < offset+numPhotons; i++)
                    {
                        const Photon & photon = photons[i];
                        float3 diff = rec.position - photon.position;
                        float distance2 = dot(diff, diff);
                        if(validPhoton(photon, distance2, radius2, rec.normal))
                        {
                            indirectAccumulatedPower += photonPower(photon, distance2, radius2);
                        }
                        _dPhotonsVisited++;
                    }

                }
            }
        }


    }

    float3 indirectRadiance = indirectAccumulatedPower * rec.attenuation * (1.0f/(M_PIf*ppmRadiusSquared)) *  (1.0f/emittedPhotonsPerIterationFloat);

    indirectRadianceBuffer[launchIndex] = indirectRadiance;
}