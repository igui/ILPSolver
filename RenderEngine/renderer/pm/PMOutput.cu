#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2014 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtBuffer<float3, 2> outputBuffer;
rtBuffer<float3, 2> indirectRadianceBuffer;
rtBuffer<float3, 2> directRadianceBuffer;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );

RT_PROGRAM void kernel()
{
    float3 finalRadiance = directRadianceBuffer[launchIndex] + indirectRadianceBuffer[launchIndex];
	outputBuffer[launchIndex] = finalRadiance;
}