#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "renderer/Hitpoint.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/helpers.h"
#include "renderer/RayType.h"
#include "renderer/RadiancePRD.h"
#include "renderer/ppm/PhotonPRD.h"
#include "renderer/ShadowPRD.h"

using namespace optix;

//
// Scene wide variables
//

rtDeclareVariable(rtObject, sceneRootObject, , );

//
// Ray generation program
//

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );

//
// Closest hit material
//

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );



RT_PROGRAM void closestHitRadiance()
{
	float3 worldShadingNormal = rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal);
	if (hitFromOutside(ray.direction, worldShadingNormal))
	{
		++radiancePrd.inHole;
	}
	else
	{
		--radiancePrd.inHole;
	}

	if(radiancePrd.depth <= MAX_RADIANCE_TRACE_DEPTH)
    {
		float3 hitPoint = ray.origin + tHit*ray.direction;
		Ray newRay = Ray(hitPoint, ray.direction, ray.ray_type, 0.0001);
        rtTrace( sceneRootObject, newRay, radiancePrd );
    }
}

/*
// Pass the photon along its way through the glass
*/

rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );

RT_PROGRAM void closestHitPhoton()
{
	float3 worldShadingNormal = rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal);
	if (hitFromOutside(ray.direction, worldShadingNormal))
	{
		++photonPrd.inHole;
	}
	else
	{
		--photonPrd.inHole;
	}

	photonPrd.depth++;
    if (photonPrd.depth <= MAX_PHOTON_TRACE_DEPTH)
    {
		float3 hitPoint = ray.origin + tHit*ray.direction;
		Ray newRay(hitPoint, ray.direction, ray.ray_type, 0.0001);
        rtTrace(sceneRootObject, newRay, photonPrd);
    }
}

rtDeclareVariable(ShadowPRD, shadowPrd, rtPayload, );

RT_PROGRAM void closestHitShadow()
{
	float3 worldShadingNormal = rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal);
	if(hitFromOutside(ray.direction, worldShadingNormal))
	{
		++shadowPrd.inHole;
	}
	else
	{
		--shadowPrd.inHole;
	}

	float3 hitPoint = ray.origin + tHit*ray.direction;
	Ray newRay(hitPoint, ray.direction, ray.ray_type, 0.0001);
    rtTrace(sceneRootObject, newRay, shadowPrd);
}