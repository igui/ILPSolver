#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "renderer/Hitpoint.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/helpers.h"
#include "renderer/RayType.h"
#include "renderer/RadiancePRD.h"
#include "renderer/ppm/PhotonPRD.h"
#include "renderer/ShadowPRD.h"

using namespace optix;

//
// Scene wide variables
//

rtDeclareVariable(rtObject, sceneRootObject, , );

//
// Ray generation program
//

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );

//
// Closest hit material
//

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );



RT_PROGRAM void closestHitRadiance()
{
	radiancePrd.flags ^= PRD_IN_HOLE;

    if(radiancePrd.depth <= MAX_RADIANCE_TRACE_DEPTH)
    {
		float3 hitPoint = ray.origin + tHit*ray.direction;
		Ray newRay = Ray(hitPoint, ray.direction, ray.ray_type, 0.0001);
        rtTrace( sceneRootObject, newRay, radiancePrd );
    }
}

/*
// Pass the photon along its way through the glass
*/

rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );

RT_PROGRAM void closestHitPhoton()
{
	photonPrd.inHole = !photonPrd.inHole;

	photonPrd.depth++;
    if (photonPrd.depth <= MAX_PHOTON_TRACE_DEPTH)
    {
		float3 hitPoint = ray.origin + tHit*ray.direction;
		Ray newRay(hitPoint, ray.direction, ray.ray_type, 0.0001);
        rtTrace(sceneRootObject, newRay, photonPrd);
    }
}

rtDeclareVariable(ShadowPRD, shadowPrd, rtPayload, );

RT_PROGRAM void closestHitShadow()
{
	shadowPrd.inHole = !shadowPrd.inHole;

	float3 hitPoint = ray.origin + tHit*ray.direction;
	Ray newRay(hitPoint, ray.direction, ray.ray_type, 0.0001);
    rtTrace(sceneRootObject, newRay, shadowPrd);
}