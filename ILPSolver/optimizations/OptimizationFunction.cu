#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_world.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include "renderer/helpers/optix.h"
#include "renderer/helpers/nsight.h"
#include "OptimizationFunction.h"
#include "renderer/PMOptixRenderer.h"
#include <QImage>

__device__ inline static optix::float3 max(optix::float3 a, optix::float3 b)
{
	 return optix::make_float3(
				a.x > b.x ? a.x : b.x,
                a.y > b.y ? a.y : b.y,
                a.z > b.z ? a.z : b.z
			);
}

__device__ inline static optix::float3 min(optix::float3 a, optix::float3 b)
{
	 return optix::make_float3(
				a.x < b.x ? a.x : b.x,
                a.y < b.y ? a.y : b.y,
                a.z < b.z ? a.z : b.z
			);
}

__global__ void transformFloatToRGB(optix::float3 *floatColorBuffer, optix::uchar3 *byteColorBuffer, float invGammaCorrection, int width, int height)
{
	unsigned int srcIndex = blockIdx.x*blockDim.x + threadIdx.x;

	auto colorFloat = floatColorBuffer[srcIndex];
	colorFloat.x = powf(colorFloat.x, invGammaCorrection);
	colorFloat.y = powf(colorFloat.y, invGammaCorrection);
	colorFloat.z = powf(colorFloat.z, invGammaCorrection);
	auto colorFloatCropped = max(
		optix::make_float3(0.0f),
		min(optix::make_float3(1.0f) , colorFloat)
	);

	optix::uchar3 colorByte;
	colorByte.x = floor(colorFloatCropped.x == 1.0f ? 255 : colorFloatCropped.x * 256.0f);
	colorByte.y = floor(colorFloatCropped.y == 1.0f ? 255 : colorFloatCropped.y * 256.0f);
	colorByte.z = floor(colorFloatCropped.z == 1.0f ? 255 : colorFloatCropped.z * 256.0f);
	
	auto x = srcIndex / width;
	auto y = srcIndex % width;
	x = height - 1  - x;
	auto dstIndex = x * width + y;

	byteColorBuffer[dstIndex] = colorByte;
}

void OptimizationFunction::saveImage(const QString& fileName)
{
	// convert float3 image data to 24 bit RGB
	nvtxRangePushA("convertImageToRGB");
    int deviceNumber = 0;
	hipSetDevice(m_renderer->deviceOrdinal());

	auto rendererOutputBuffer = m_renderer->outputBuffer();
	auto floatOutputBuffer = getThrustDevicePtr<optix::float3>(rendererOutputBuffer, deviceNumber);
	auto floatOutputBufferPtr = thrust::raw_pointer_cast(&floatOutputBuffer[0]);

	int imageSize = sampleImageWidth * sampleImageHeight;
	thrust::device_vector<optix::uchar3> byteOutputBuffer(imageSize);
	auto byteOutputBufferPtr = thrust::raw_pointer_cast(&byteOutputBuffer[0]);

	const unsigned int blockSize = 512;
    unsigned int numBlocks = imageSize/blockSize + (imageSize % blockSize == 0 ? 0 : 1);

	transformFloatToRGB<<<numBlocks, blockSize>>> (floatOutputBufferPtr, byteOutputBufferPtr, 1.0f / gammaCorrection, sampleImageWidth, sampleImageHeight);

	hipDeviceSynchronize();
	nvtxRangePop();
	
	// read image data and write it in an image object
	optix::uchar3 *imageBytes = new optix::uchar3[imageSize];
	thrust::copy(byteOutputBuffer.begin(), byteOutputBuffer.end(), imageBytes);
	auto image = new QImage((uchar *) imageBytes, sampleImageWidth, sampleImageHeight, QImage::Format_RGB888);
	
	// save image to a temporary file
	saveImageAsync(fileName, image);
}