#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_world.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include "renderer/helpers/optix.h"
#include "renderer/helpers/nsight.h"
#include "SurfaceRadiosity.h"
#include "renderer/PMOptixRenderer.h"
#include <QImage>

__device__ inline static optix::float3 max(optix::float3 a, optix::float3 b)
{
	 return optix::make_float3(
				a.x > b.x ? a.x : b.x,
                a.y > b.y ? a.y : b.y,
                a.z > b.z ? a.z : b.z
			);
}

__device__ inline static optix::float3 min(optix::float3 a, optix::float3 b)
{
	 return optix::make_float3(
				a.x < b.x ? a.x : b.x,
                a.y < b.y ? a.y : b.y,
                a.z < b.z ? a.z : b.z
			);
}

__global__ void transformFloatToRGB(optix::float3 *floatColorBuffer, optix::uchar3 *byteColorBuffer)
{
	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

	auto colorFloat = floatColorBuffer[index];
	auto colorFloatCropped = max(
		optix::make_float3(0.0f),
		min(optix::make_float3(1.0f) , colorFloat)
	);

	optix::uchar3 colorByte;
	colorByte.x = floor(colorFloatCropped.x == 1.0f ? 255 : colorFloatCropped.x * 256.0f);
	colorByte.y = floor(colorFloatCropped.y == 1.0f ? 255 : colorFloatCropped.y * 256.0f);
	colorByte.z = floor(colorFloatCropped.z == 1.0f ? 255 : colorFloatCropped.z * 256.0f);
	byteColorBuffer[index] = colorByte;
}

void SurfaceRadiosity::saveImage()
{
	nvtxRangePushA("convertImageToRGB");
    int deviceNumber = 0;
	hipSetDevice(renderer->deviceOrdinal());

	auto rendererOutputBuffer = renderer->outputBuffer();
	auto floatOutputBuffer = getThrustDevicePtr<optix::float3>(rendererOutputBuffer, deviceNumber);
	auto floatOutputBufferPtr = thrust::raw_pointer_cast(&floatOutputBuffer[0]);

	thrust::device_vector<optix::uchar3> byteOutputBuffer(sampleImageWidth * sampleImageHeight);
	auto byteOutputBufferPtr = thrust::raw_pointer_cast(&byteOutputBuffer[0]);

	const unsigned int blockSize = 512;
    unsigned int numBlocks = (sampleImageWidth * sampleImageHeight)/blockSize + ((sampleImageWidth * sampleImageHeight) % blockSize == 0 ? 0 : 1);

	transformFloatToRGB<<<numBlocks, blockSize>>> (floatOutputBufferPtr, byteOutputBufferPtr);

	hipDeviceSynchronize();
	nvtxRangePop();
		
	optix::uchar3 *imageBytes = new optix::uchar3[sampleImageWidth * sampleImageHeight];
	thrust::copy(byteOutputBuffer.begin(), byteOutputBuffer.end(), imageBytes);
	auto image = QImage((uchar *) imageBytes, sampleImageWidth, sampleImageHeight, QImage::Format_RGB888);
	image.mirrored(false, true).save("C:\\Users\\Igui\\Desktop\\image.png");
}